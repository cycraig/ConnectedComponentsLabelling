
#include <hip/hip_runtime.h>
#include <helper_functions.h>    // includes hip/hip_runtime.h and hip/hip_runtime_api.h
#include <hip/hip_runtime_api.h>
#include <cstring> 
#include <cmath>
#include "hip/hip_runtime.h"
#include "book.h" //HANDLE_ERROR
#include "cpu_bitmap.h" 

#include "EasyBMP.h"

// globals needed by the update routine
struct DataBlock {
    unsigned char   *output_bitmap;
    float           *dev_inSrc;
    float           *dev_outSrc;
    float           *dev_constSrc;
    //CPUAnimBitmap  *bitmap;
    CPUBitmap  *bitmap;

    hipEvent_t     start, stop;
    float           totalTime;
    float           frames;
};

void anim_gpu( DataBlock *d, int ticks ) {
    /*HANDLE_ERROR( hipEventRecord( d->start, 0 ) );
    dim3    blocks(DIM/16,DIM/16);
    dim3    threads(16,16);
    CPUAnimBitmap  *bitmap = d->bitmap;

    // since tex is global and bound, we have to use a flag to
    // select which is in/out per iteration
    volatile bool dstOut = true;
    for (int i=0; i<90; i++) {
        float   *in, *out;
        if (dstOut) {
            in  = d->dev_inSrc;
            out = d->dev_outSrc;
        } else {
            out = d->dev_inSrc;
            in  = d->dev_outSrc;
        }
        copy_const_kernel<<<blocks,threads>>>( in );
        blend_kernel<<<blocks,threads>>>( out, dstOut );
        dstOut = !dstOut;
    }
    float_to_color<<<blocks,threads>>>( d->output_bitmap,
                                        d->dev_inSrc );

    HANDLE_ERROR( hipMemcpy( bitmap->get_ptr(),
                              d->output_bitmap,
                              bitmap->image_size(),
                              hipMemcpyDeviceToHost ) );

    HANDLE_ERROR( hipEventRecord( d->stop, 0 ) );
    HANDLE_ERROR( hipEventSynchronize( d->stop ) );
    float   elapsedTime;
    HANDLE_ERROR( hipEventElapsedTime( &elapsedTime,
                                        d->start, d->stop ) );
    d->totalTime += elapsedTime;
    ++d->frames;
    printf( "Average Time per frame:  %3.1f ms\n",
            d->totalTime/d->frames  );*/
}

// clean up memory allocated on the GPU
void anim_exit( DataBlock *d ) {
    /*
    hipUnbindTexture( texIn );
    hipUnbindTexture( texOut );
    hipUnbindTexture( texConstSrc );
    HANDLE_ERROR( hipFree( d->dev_inSrc ) );
    HANDLE_ERROR( hipFree( d->dev_outSrc ) );
    HANDLE_ERROR( hipFree( d->dev_constSrc ) );

    HANDLE_ERROR( hipEventDestroy( d->start ) );
    HANDLE_ERROR( hipEventDestroy( d->stop ) );
    */
}

void copyBMPtoBitmap(BMP* input, CPUBitmap* output) {
	unsigned char *rgbaPixels = output->get_ptr();
	int width = input->TellWidth();
	int height = input->TellHeight();
	for(int y = 0; y < height; y++) {
		for(int x = 0; x < width; x++) {
			rgbaPixels[y*4*width+4*x]   = (*input)(x,y)->Red;
			rgbaPixels[y*4*width+4*x+1] = (*input)(x,y)->Green;
			rgbaPixels[y*4*width+4*x+2] = (*input)(x,y)->Blue;
			rgbaPixels[y*4*width+4*x+3] = (*input)(x,y)->Alpha;
		}
	}
}

void copyBitmapToBMP(CPUBitmap* input, BMP* output) {
	unsigned char *rgbaPixels = input->get_ptr();
	int width = input->x;
	int height = input->y;
	output->SetSize(width,height);
	for(int y = 0; y < height; y++) {
		for(int x = 0; x < width; x++) {
			(*output)(x,y)->Red   = rgbaPixels[y*4*width+4*x];
		    (*output)(x,y)->Green = rgbaPixels[y*4*width+4*x+1];
			(*output)(x,y)->Blue  = rgbaPixels[y*4*width+4*x+2];
			(*output)(x,y)->Alpha = rgbaPixels[y*4*width+4*x+3];
		}
	}
}

void bitmapToBinary(CPUBitmap* input, int *output) {
	unsigned char *rgbaPixels = input->get_ptr();
	int width = input->x;
	int height = input->y;
	// output should be of size width*height
	// assuming 4 byte stride for RGBA values
	for(int y = 0; y < height; y++) {
		for(int x = 0; x < width; x++) {
			unsigned char r = rgbaPixels[y*4*width+4*x];
			unsigned char g = rgbaPixels[y*4*width+4*x+1];
			unsigned char b = rgbaPixels[y*4*width+4*x+2];
			// Thresholding according to: (r+g+b)/3 > 128
			output[y*width+x] = ((r+g+b) > 348); // 1 -> white, 0 -> black
		}
	}
}

void binaryToBitmap(int *input, CPUBitmap* output) {
	unsigned char *rgbaPixels = output->get_ptr();
	int width = output->x;
	int height = output->y;
	// assuming 4 byte stride for RGBA values
	for(int y = 0; y < height; y++) {
		for(int x = 0; x < width; x++) {
			// [0,1] -> [0,255]
			rgbaPixels[y*4*width+4*x]   = input[y*width+x]*255;
			rgbaPixels[y*4*width+4*x+1] = input[y*width+x]*255;
			rgbaPixels[y*4*width+4*x+2] = input[y*width+x]*255;
			rgbaPixels[y*4*width+4*x+3] = 255;
		}
	}
}

void colourise(int* input, CPUBitmap* output, int** labelColours) {
	unsigned char *rgbaPixels = output->get_ptr();
	int width = output->x;
	int height = output->y;
	for(int y = 0; y < height; y++) {
		for(int x = 0; x < width; x++) {
			int label = input[y*width+x];
			if(label == 0) {
				rgbaPixels[y*4*width+4*x]   = 0;
				rgbaPixels[y*4*width+4*x+1] = 0;
				rgbaPixels[y*4*width+4*x+2] = 0;
				rgbaPixels[y*4*width+4*x+3] = 255;
				continue;
			}

			int r = labelColours[label][0];
			int g = labelColours[label][1];
			int b = labelColours[label][2];
			rgbaPixels[y*4*width+4*x]   = r;
			rgbaPixels[y*4*width+4*x+1] = g;
			rgbaPixels[y*4*width+4*x+2] = b;
			rgbaPixels[y*4*width+4*x+3] = 255;
		}
	}
}

void getLabelColours(int** labelColours, int maxLabels) {
	for(int i = 1; i <= maxLabels; i++) {
		labelColours[i][0] = i * 131 % 255;
		labelColours[i][1] = i * 241 % 255;
		labelColours[i][2] = i * 251 % 255;
	}
}

void markEquivalent(int** equivalenceMatrix, int a, int b) {
	equivalenceMatrix[a][b] = 1;
	equivalenceMatrix[b][a] = 1;
}

void printMatrix(int** matrix, int width, int height) {
	for(int y = 0; y < height; y++) {
		for(int x = 0; x < width; x++) {
			int val = matrix[y][x];
			if(val < 10) {
				printf(" %d ",val);
			} else {
				printf("%d ",val);
			}
		}
		printf("\n");
	}
}

void printArray(int* array, int size) {
	for(int i = 0; i < size; i++) {
		printf("%d ",array[i]);
	}
	printf("\n");
}

void updateLabelArray(int* labelArray, int** L, int maxLabel) {
	labelArray[0] = 0;
	for(int label = 1; label <= maxLabel; label++) {
		for(int i = 1; i <= maxLabel; i++) {
			if(L[label][i]) {
				labelArray[label] = i;
				break;
			}
		}
	}
	printf("LABEL ARRAY:\n");
	printArray(labelArray,maxLabel+1);
}

void resolveEquivalences(int** L, int maxLabel) {
	int n = maxLabel;
	for(int j = 1; j <= n; j++) {
		for(int i = 1; i <= n; i++) {
			if(L[i][j] == 1) {
				for(int k = 1; k <= n; k++) {
					L[i][k] = L[i][k] || L[j][k];
				}
			}
		}
	}
	//printf("EQUIVALENCE MATRIX:\n");
	//printMatrix(L,n+1,n+1);
}

void updateRegion(int* region, int* labelArray, int width, int height) {
	for(int y = 0; y < height; y++) {
		for(int x = 0; x < width; x++) {
			int idx = y*width+x;
			if(region[idx] > 0) {
				region[idx] = labelArray[region[idx]];
			}
		}
	}
}

void printLabels(int* region, int width, int height) {
	for(int y = 0; y < height; y++) {
		for(int x = 0; x < width; x++) {
			int val = region[y*width+x];
			if(val < 10) {
				printf(" %d ",val);
			} else {
				printf("%d ",val);
			}
		}
		printf("\n");
	}
}

void label(int* region, CPUBitmap* output, int width, int height) {
	// assume maximum number of labels
	int size = width*height/2.0+1;
	
	int** equivalenceMatrix = new int*[size];
	for(int i = 0; i < size; i++) {
		equivalenceMatrix[i] = new int[size];
		memset(equivalenceMatrix[i], 0, sizeof(int)*size);
		// reflexivity
		equivalenceMatrix[i][i] = 1;
	}

	int labelCount = 0;

	// initial labelling
	for(int y = 0; y < height; y++) {
		for(int x = 0; x < width; x++) {
			
			// ignore background pixel
			if(region[y*width+x] == 0) continue;

			// check 4-nbrs
			int n=0,nw=0,ne=0,w=0,label=0;
			if(x > 0) {
				w = region[y*width+x-1];
				if(w) {
					label = w;
				}
			}
			if(y > 0) {
				n = region[(y-1)*width+x];
				if(n) {
					label = n;
					if(w) markEquivalent(equivalenceMatrix,n,w);
				}
			}
			if(y > 0 && x > 0) {
				nw = region[(y-1)*width+(x-1)];
				if(nw) {
					label = nw;
					if(w) markEquivalent(equivalenceMatrix,nw,w);
					if(n) markEquivalent(equivalenceMatrix,nw,n);
				}
			}
			if(y > 0 && x < (width-1)) {
				ne = region[(y-1)*width+(x+1)];
				if(ne) {
					label = ne;
					if(w)  markEquivalent(equivalenceMatrix,ne,w);
					if(n)  markEquivalent(equivalenceMatrix,ne,n);
					if(nw) markEquivalent(equivalenceMatrix,ne,nw);
				}
			}
			if(label == 0) {
				labelCount++;
				label = labelCount;
			}
			region[y*width+x] = label;
		}
	}
	resolveEquivalences(equivalenceMatrix,labelCount);
	int* labelArray = new int[labelCount+1];
	updateLabelArray(labelArray,equivalenceMatrix,labelCount);
	printf("BEFORE:\n");
	printLabels(region,width,height);
	updateRegion(region,labelArray,width,height);
	printf("AFTER:\n");
	printLabels(region,width,height);
	
	int** labelColours = new int*[labelCount+1];
	for(int i = 1; i < labelCount+1; i++) labelColours[i] = new int[3];
	getLabelColours(labelColours,labelCount);
	colourise(region,output,labelColours);
}

int main(int argc, char **argv) {
    printf("%s Starting...\n\n", argv[0]);

    //initialize CUDA
    findCudaDevice(argc, (const char **)argv);

    //source and results image filenames
    char SampleImageFname[] = "sections2.bmp";
    char *pSampleImageFpath = sdkFindFilePath(SampleImageFname, argv[0]);

    if (pSampleImageFpath == NULL) {
        printf("%s could not locate Sample Image <%s>\nExiting...\n", pSampleImageFpath);
        exit(EXIT_FAILURE);
    }

    BMP input;
    BMP output;

	printf("===================================\n");
    printf("Loading image: %s... ", pSampleImageFpath);
    bool result = input.ReadFromFile(pSampleImageFpath);
	if (result == false) {
        printf("\nError: Image file not found or invalid!\n");
        exit(EXIT_FAILURE);
        return 1;
    }
	int width = input.TellWidth();
	int height = input.TellHeight();
	output.SetSize(width,height);
	output.SetBitDepth(32); // RGBA
    DataBlock   data;
    CPUBitmap bitmap( width, height, &data );
    data.bitmap = &bitmap;
    data.totalTime = 0;
    data.frames = 0;
    //HANDLE_ERROR( hipEventCreate( &data.start ) );
    //HANDLE_ERROR( hipEventCreate( &data.stop ) );
	copyBMPtoBitmap(&input,&bitmap);
	int* binaryImage = new int[width*height];
	bitmapToBinary(&bitmap,binaryImage);
	printf("LABELLING...\n");
	label(binaryImage,&bitmap,width,height);
	printf("FINISHED...\n");
	copyBitmapToBMP(&bitmap,&output);
    //HANDLE_ERROR( hipMemcpy( bitmap.get_ptr(), ImgSrc, imageSize, hipMemcpyHostToHost ) );
    //DumpBmpAsGray("out.bmp", ImgSrc, ImgStride, ImgSize);
    output.WriteToFile("out.bmp");
    bitmap.display_and_exit((void (*)(void*))anim_exit);
}

