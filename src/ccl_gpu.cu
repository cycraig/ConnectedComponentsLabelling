#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "../inc/hip/hip_runtime_api.h"
#include "common_ccl.h"

#define cudaErrorCheck(t) { \
 t; \
 hipError_t e=hipGetLastError(); \
 if(e!=hipSuccess) { \
   printf("Cuda failure %s:%d: '%s'\n",__FILE__,__LINE__,hipGetErrorString(e)); \
   exit(0); \
 } \
}

int regionWidth = 8;
int regionHeight = 8;
int total_index;

//Texture binding variable
surface<void, cudaSurfaceType2D> surf_ref;

__global__ void gpu_label(int width, int height) {
    // STEP 1 - Initial Labelling

	int x = blockIdx.x*blockDim.x + threadIdx.x;
	int y = blockIdx.y*blockDim.y + threadIdx.y;
    int idx = width*y+x+1; // +1 to avoid 0 labels

	int temp;
    if ((x<width) && (y<height)) {
        surf2Dread(&temp, surf_ref, x*sizeof(int), y, hipBoundaryModeZero);
        if(temp != 0) {
	        surf2Dwrite(idx, surf_ref, x*sizeof(int), y, hipBoundaryModeZero);
        }
	}
}

__device__ int getMinNeighbourScan(int x, int y, int label) {
    // boundary mode zero causes out of range reads to return 0 (convenient)
    int minLabel = label, curr = -1;
    // south-west
    surf2Dread(&curr, surf_ref, (x-1)*sizeof(int), y+1, hipBoundaryModeZero);
    if(curr > 0) minLabel = min(minLabel,curr);
    // west
    surf2Dread(&curr, surf_ref, (x-1)*sizeof(int), y, hipBoundaryModeZero);
    if(curr > 0) minLabel = min(minLabel,curr);
    // north-west
    surf2Dread(&curr, surf_ref, (x-1)*sizeof(int), y-1, hipBoundaryModeZero);
    if(curr > 0) minLabel = min(minLabel,curr);
    // north
    surf2Dread(&curr, surf_ref, x*sizeof(int), y-1, hipBoundaryModeZero);
    if(curr > 0) minLabel = min(minLabel,curr);
    // north-east
    surf2Dread(&curr, surf_ref, (x+1)*sizeof(int), y-1, hipBoundaryModeZero);
    if(curr > 0) minLabel = min(minLabel,curr);
    return minLabel;
}

__global__ void gpu_scan(int width, int height) {
    // STEP 2 - Scanning
	int x = blockIdx.x*blockDim.x + threadIdx.x;
	int y = blockIdx.y*blockDim.y + threadIdx.y;

	int label;
    if ((x<width) && (y<height)) {
	    surf2Dread(&label, surf_ref, x*sizeof(int), y, hipBoundaryModeZero);
        if(label != 0) {
            int minLabelScanned = getMinNeighbourScan(x,y,label);
	        surf2Dwrite(minLabelScanned, surf_ref, x*sizeof(int), y, hipBoundaryModeZero);
        }
	}
}

__global__ void gpu_analysis(int width, int height) {
    // STEP 3 - Analysis
	int x = blockIdx.x*blockDim.x + threadIdx.x;
	int y = blockIdx.y*blockDim.y + threadIdx.y;

	int label;
    if ((x<width) && (y<height)) {
	    surf2Dread(&label, surf_ref, x*sizeof(int), y, hipBoundaryModeZero);
        if(label != 0) {
            // propagate labels
            // "recursively" get the final label
            // - if first referred pixel index refers to another label
            // - stop when the label refers to itself
            int idx = -1;
            int lx,ly;
            while(label != (idx+1)) {
                idx = label-1; // -1 since labels start from 1 and we want 1D pixel index
                lx = idx%width;
                ly = idx/width;
                surf2Dread(&label, surf_ref, lx*sizeof(int), ly, hipBoundaryModeZero);
            }
	        surf2Dwrite(label, surf_ref, x*sizeof(int), y, hipBoundaryModeZero);
        }
	}
}

__device__ int getMinNeighbourLink(int x, int y, int label) {
    // boundary mode zero causes out of range reads to return 0 (convenient)
    int minLabel = label, curr = -1;

	// CHANGED FROM PAPER
	// Need to check south-east, north, and north-west as well for the algorithm to work

    // south-west
    surf2Dread(&curr, surf_ref, (x-1)*sizeof(int), y+1, hipBoundaryModeZero);
    if(curr > 0) minLabel = min(minLabel,curr);
	// south-east
	surf2Dread(&curr, surf_ref, (x+1)*sizeof(int), y+1, hipBoundaryModeZero);
    if(curr > 0) minLabel = min(minLabel,curr);
    // west
    surf2Dread(&curr, surf_ref, (x-1)*sizeof(int), y, hipBoundaryModeZero);
    if(curr > 0) minLabel = min(minLabel,curr);
    // east
    surf2Dread(&curr, surf_ref, (x+1)*sizeof(int), y, hipBoundaryModeZero);
    if(curr > 0) minLabel = min(minLabel,curr);
    // north-east
    surf2Dread(&curr, surf_ref, (x+1)*sizeof(int), y-1, hipBoundaryModeZero);
    if(curr > 0) minLabel = min(minLabel,curr);
	// north
    surf2Dread(&curr, surf_ref, x*sizeof(int), y-1, hipBoundaryModeZero);
    if(curr > 0) minLabel = min(minLabel,curr);
	// north-west
    surf2Dread(&curr, surf_ref, (x-1)*sizeof(int), y-1, hipBoundaryModeZero);
    if(curr > 0) minLabel = min(minLabel,curr);
    return minLabel;
}

__global__ void gpu_link(int width, int height) {
    // STEP 4 - Link
	int x = blockIdx.x*blockDim.x + threadIdx.x;
	int y = blockIdx.y*blockDim.y + threadIdx.y;

	int label;
    if ((x<width) && (y<height)) {
	    surf2Dread(&label, surf_ref, x*sizeof(int), y, hipBoundaryModeZero);
        if(label != 0) {
            // scan neighbours
            int minLabel = getMinNeighbourLink(x,y,label);
            // update pixel of REFERENCE label (not current pixel)
            // this is so that all other pixels can simply reference that pixel
            // in the next step
            if(minLabel < label) {
                int refIdx = label-1; // -1 since labels start from 1 and we want 1D pixel index
                int refx = refIdx%width;
                int refy = refIdx/width;
	            surf2Dwrite(minLabel, surf_ref, refx*sizeof(int), refy, hipBoundaryModeZero);
            }
        }
	}
}

__global__ void gpu_relabel(int width, int height) {
    // STEP 5 - Re-label
	int x = blockIdx.x*blockDim.x + threadIdx.x;
	int y = blockIdx.y*blockDim.y + threadIdx.y;

	int label;
    if ((x<width) && (y<height)) {
	    surf2Dread(&label, surf_ref, x*sizeof(int), y, hipBoundaryModeZero);
        if(label != 0) {
            // resolve label equivalences (after previous step)
            int refIdx = label-1; // -1 since labels start from 1 and we want 1D pixel index
            int refx = refIdx%width;
            int refy = refIdx/width;
            int refLabel;
            surf2Dread(&refLabel, surf_ref, refx*sizeof(int), refy, hipBoundaryModeZero);
            surf2Dwrite(refLabel, surf_ref, x*sizeof(int), y, hipBoundaryModeZero);
        }
	}
}

__device__ bool done;
__global__ void gpu_rescan(int width, int height) {
    // STEP 5 - Re-Scan
	int x = blockIdx.x*blockDim.x + threadIdx.x;
	int y = blockIdx.y*blockDim.y + threadIdx.y;

	int label;
    if ((x<width) && (y<height)) {
	    surf2Dread(&label, surf_ref, x*sizeof(int), y, hipBoundaryModeZero);
        if(label != 0) {
            // check if all regions are connected
            int minNeighbour = getMinNeighbourScan(x,y,label);
            if(minNeighbour != label) {
                done = false;
            }
        }
	}
}

void gpu_label(int* image, CPUBitmap* output, int width, int height, float* gpuTime) {
    hipArray* gpuImage;
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindSigned);
    cudaErrorCheck(hipMallocArray(&gpuImage, &channelDesc, width, height, hipArraySurfaceLoadStore));
    cudaErrorCheck(hipMemcpyToArray(gpuImage, 0, 0, image, width*height*sizeof(int), hipMemcpyHostToDevice));
    cudaErrorCheck(cudaBindSurfaceToArray(surf_ref, gpuImage));

    dim3 block_dim(regionWidth, regionHeight);
    int gridWidth = width/block_dim.x;
    int gridHeight = height/block_dim.y;
    if (width%block_dim.x != 0) gridWidth++;
    if (height%block_dim.y != 0) gridHeight++;
    int result = false;
    dim3 grid_dim(gridWidth, gridHeight);
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    gpu_label<<<grid_dim, block_dim>>>(width, height);
    gpu_scan<<<grid_dim, block_dim>>>(width, height);
    gpu_analysis<<<grid_dim, block_dim>>>(width, height);
    while(result == false) {
        gpu_link<<<grid_dim, block_dim>>>(width, height);
        gpu_relabel<<<grid_dim, block_dim>>>(width, height);
        result = true;
        cudaErrorCheck(hipMemcpyToSymbol(HIP_SYMBOL(done), &result, sizeof(bool)));
        gpu_rescan<<<grid_dim, block_dim>>>(width, height);
        cudaErrorCheck(hipMemcpyFromSymbol(&result, HIP_SYMBOL(done), sizeof(bool)));
    }

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    *gpuTime = 0;
    hipEventElapsedTime(gpuTime, start, stop);
    cudaErrorCheck(hipMemcpyFromArray(image, gpuImage, 0, 0,width*height*sizeof(int), hipMemcpyDeviceToHost));
	// apparently you don't need to unbind surfaces
    cudaErrorCheck(hipFreeArray(gpuImage));
}

int main(int argc, char **argv) {
  int width, height;
	int* dims = new int[2];
	int* binaryImage;
	CPUBitmap *bitmap;
	DataBlock data;
	BMP output;
  BMP input;
  struct arguments parsed_args;

  if (!start(argc, argv,
      width, height,
      input,
      parsed_args)) exit(EXIT_FAILURE);

  regionWidth = parsed_args.region_width;
  regionHeight = parsed_args.region_width;

  bitmap = new CPUBitmap( width, height, &data );
  data.bitmap = bitmap;
  copyBMPtoBitmap(&input,bitmap);
  binaryImage = new int[(width)*(height)];
  bitmapToBinary(bitmap,binaryImage);
  output.SetSize(width,height);
  output.SetBitDepth(32); // RGBA

    fprintf(stderr,"LABELLING...\n");

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    float gpuTime = 0;
    gpu_label(binaryImage,bitmap,width,height,&gpuTime);

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    fprintf(stderr,"FINISHED...\n");

    if (!parsed_args.bench) {
      printf("Time elapsed (gpu): %.6f ms\n",gpuTime);
      printf("Time elapsed (total): %.6f ms\n",milliseconds);
    }
    else {
      printf("%s,%d,%f,%f\n",parsed_args.mode==NORMAL_MODE?"normal":"random",
       width*height,
       gpuTime,milliseconds);
    }

    finish(width, height,
            output,
            bitmap,
            binaryImage,
            parsed_args,
            "ccl_gpu");
    delete[] binaryImage;
}
