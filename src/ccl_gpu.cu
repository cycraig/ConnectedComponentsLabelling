#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "../inc/hip/hip_runtime_api.h"
#include "common_ccl.h"

#define cudaErrorCheck(t) { \
 t; \
 hipError_t e=hipGetLastError(); \
 if(e!=hipSuccess) { \
   printf("Cuda failure %s:%d: '%s'\n",__FILE__,__LINE__,hipGetErrorString(e)); \
   exit(0); \
 } \
}

int regionWidth = 8;
int regionHeight = 8;
int total_index;

//Texture binding variable
surface<void, cudaSurfaceType2D> surf_ref; 

void colourise(int* input, CPUBitmap* output, int width, int height) {
	unsigned char *rgbaPixels = output->get_ptr();
	for(int y = 0; y < height; y++) {
		for(int x = 0; x < width; x++) {
			int label = input[y*width+x];
			if(label == 0) {
				rgbaPixels[y*4*width+4*x]   = 0;
				rgbaPixels[y*4*width+4*x+1] = 0;
				rgbaPixels[y*4*width+4*x+2] = 0;
				rgbaPixels[y*4*width+4*x+3] = 255;
				continue;
			}
			rgbaPixels[y*4*width+4*x]   = input[y*width+x] * 131 % 255;
			rgbaPixels[y*4*width+4*x+1] = input[y*width+x] * 241 % 255;
			rgbaPixels[y*4*width+4*x+2] = input[y*width+x] * 251 % 255;
			rgbaPixels[y*4*width+4*x+3] = 255;
		}
	}
}

__global__ void gpu_label(int width, int height) {
    // STEP 1 - Initial Labelling

	//From https://cs.calvin.edu/courses/cs/374/CUDA/CUDA-Thread-Indexing-Cheatsheet.pdf
	//int i = (blockIdx.x + blockIdx.y * gridDim.x) * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;
	int x = blockIdx.x*blockDim.x + threadIdx.x;
	int y = blockIdx.y*blockDim.y + threadIdx.y;
    int idx = width*y+x+1; // +1 to avoid 0 labels

	int temp;
    if ((x<width) && (y<height)) {
        surf2Dread(&temp, surf_ref, x*sizeof(int), y, hipBoundaryModeZero);
        if(temp != 0) {
	        surf2Dwrite(idx, surf_ref, x*sizeof(int), y, hipBoundaryModeZero);
        }
        //printf("x = %d, y = %d, i = %d\n",x,y,idx);
	}
}

__device__ int getMinNeighbourScan(int x, int y, int label) {
    // boundary mode zero causes out of range reads to return 0 (convenient)
    int minLabel = label, curr;
    // south-west
    surf2Dread(&curr, surf_ref, (x-1)*sizeof(int), y+1, hipBoundaryModeZero);
    if(curr > 0) minLabel = min(minLabel,curr);
    // west
    surf2Dread(&curr, surf_ref, (x-1)*sizeof(int), y, hipBoundaryModeZero);
    if(curr > 0) minLabel = min(minLabel,curr);
    // north-west
    surf2Dread(&curr, surf_ref, (x-1)*sizeof(int), y-1, hipBoundaryModeZero);
    if(curr > 0) minLabel = min(minLabel,curr);
    // north
    surf2Dread(&curr, surf_ref, x*sizeof(int), y-1, hipBoundaryModeZero);
    if(curr > 0) minLabel = min(minLabel,curr);
    // north-east
    surf2Dread(&curr, surf_ref, (x+1)*sizeof(int), y-1, hipBoundaryModeZero);
    if(curr > 0) minLabel = min(minLabel,curr);
    return minLabel;
}

__global__ void gpu_scan(int width, int height) {
    // STEP 2 - Scanning
	int x = blockIdx.x*blockDim.x + threadIdx.x;
	int y = blockIdx.y*blockDim.y + threadIdx.y;

	int label;
    if ((x<width) && (y<height)) {
	    surf2Dread(&label, surf_ref, x*sizeof(int), y, hipBoundaryModeZero);
        if(label != 0) {
            int minLabelScanned = getMinNeighbourScan(x,y,label);
	        surf2Dwrite(minLabelScanned, surf_ref, x*sizeof(int), y, hipBoundaryModeZero);
        }
	}
}

__global__ void gpu_analysis(int width, int height) {
    // STEP 3 - Analysis
	int x = blockIdx.x*blockDim.x + threadIdx.x;
	int y = blockIdx.y*blockDim.y + threadIdx.y;
    //int i = y*width + x;

	int label;
    if ((x<width) && (y<height)) {
	    surf2Dread(&label, surf_ref, x*sizeof(int), y, hipBoundaryModeZero);
        if(label != 0) {
            // propagate labels
            // "recursively" get the final label
            // - if first referred pixel index refers to another label
            // - stop when the label refers to itself
            int idx = -1;
            int lx,ly;
            while(label != (idx+1)) {
                idx = label-1; // -1 since labels start from 1 and we want 1D pixel index
                lx = idx%width;
                ly = idx/width;
                surf2Dread(&label, surf_ref, lx*sizeof(int), ly, hipBoundaryModeZero);
            }
	        surf2Dwrite(label, surf_ref, x*sizeof(int), y, hipBoundaryModeZero);
        }
	}
}

__device__ int getMinNeighbourLink(int x, int y, int label) {
    // boundary mode zero causes out of range reads to return 0 (convenient)
    int minLabel = label, curr;
    // south-west
    surf2Dread(&curr, surf_ref, (x-1)*sizeof(int), y+1, hipBoundaryModeZero);
    if(curr > 0) minLabel = min(minLabel,curr);
    // west
    surf2Dread(&curr, surf_ref, (x-1)*sizeof(int), y, hipBoundaryModeZero);
    if(curr > 0) minLabel = min(minLabel,curr);
    // east
    surf2Dread(&curr, surf_ref, (x+1)*sizeof(int), y, hipBoundaryModeZero);
    if(curr > 0) minLabel = min(minLabel,curr);
    // north-east
    surf2Dread(&curr, surf_ref, (x+1)*sizeof(int), y-1, hipBoundaryModeZero);
    if(curr > 0) minLabel = min(minLabel,curr);
    return minLabel;
}

__global__ void gpu_link(int width, int height) {
    // STEP 4 - Link
	int x = blockIdx.x*blockDim.x + threadIdx.x;
	int y = blockIdx.y*blockDim.y + threadIdx.y;
    //int i = y*width + x;

	int label;
    if ((x<width) && (y<height)) {
	    surf2Dread(&label, surf_ref, x*sizeof(int), y, hipBoundaryModeZero);
        if(label != 0) {
            // scan neighbours
            int minLabel = getMinNeighbourLink(x,y,label);
            // update pixel of REFERENCE label (not current pixel)
            // this is so that all other pixels can simply reference that pixel
            // in the next step
            if(minLabel < label) {
                int refIdx = label-1; // -1 since labels start from 1 and we want 1D pixel index
                int refx = refIdx%width;
                int refy = refIdx/width;
	            surf2Dwrite(minLabel, surf_ref, refx*sizeof(int), refy, hipBoundaryModeZero);
            }
        }
	}
}

__global__ void gpu_relabel(int width, int height) {
    // STEP 5 - Re-label
	int x = blockIdx.x*blockDim.x + threadIdx.x;
	int y = blockIdx.y*blockDim.y + threadIdx.y;
    //int i = y*width + x;

	int label;
    if ((x<width) && (y<height)) {
	    surf2Dread(&label, surf_ref, x*sizeof(int), y, hipBoundaryModeZero);
        if(label != 0) {
            // resolve label equivalences (after previous step)
            int refIdx = label-1; // -1 since labels start from 1 and we want 1D pixel index
            int refx = refIdx%width;
            int refy = refIdx/width;
            int refLabel;
            surf2Dread(&refLabel, surf_ref, refx*sizeof(int), refy, hipBoundaryModeZero);
            surf2Dwrite(refLabel, surf_ref, x*sizeof(int), y, hipBoundaryModeZero);
        }
	}
}

__device__ bool done;
__global__ void gpu_rescan(int width, int height) {
    // STEP 5 - Re-Scan
	int x = blockIdx.x*blockDim.x + threadIdx.x;
	int y = blockIdx.y*blockDim.y + threadIdx.y;
    //int i = y*width + x;

	int label;
    if ((x<width) && (y<height)) {
	    surf2Dread(&label, surf_ref, x*sizeof(int), y, hipBoundaryModeZero);
        if(label != 0) {
            // check if all regions are connected
            int minNeighbour = getMinNeighbourScan(x,y,label);
            if(minNeighbour != label) {
                done = false;
            }
        }
	}
}

void gpu_label(int* image, CPUBitmap* output, int width, int height) {
    hipArray* gpuImage;
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindSigned);
    cudaErrorCheck(hipMallocArray(&gpuImage, &channelDesc, width, height, hipArraySurfaceLoadStore));
    cudaErrorCheck(hipMemcpyToArray(gpuImage, 0, 0, image, width*height*sizeof(int), hipMemcpyHostToDevice));
    cudaErrorCheck(cudaBindSurfaceToArray(surf_ref, gpuImage));
    

    dim3 block_dim(regionWidth, regionHeight);
    int gridWidth = width/block_dim.x;
    int gridHeight = height/block_dim.y;
    if (width%block_dim.x != 0) gridWidth++;
    if (height%block_dim.y != 0) gridHeight++;
    int result = false;
    dim3 grid_dim(gridWidth, gridHeight);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    //printf("Initial...\n");
    //printMatrix(image,width,height);
    gpu_label<<<grid_dim, block_dim>>>(width, height);
    //cudaErrorCheck(hipMemcpyFromArray(image, gpuImage, 0, 0,width*height*sizeof(int), hipMemcpyDeviceToHost));
    //printf("AFTER LABELLING...\n");
    //printMatrix(image,width,height);
    gpu_scan<<<grid_dim, block_dim>>>(width, height);
    //cudaErrorCheck(hipMemcpyFromArray(image, gpuImage, 0, 0,width*height*sizeof(int), hipMemcpyDeviceToHost));
    //printf("AFTER SCAN...\n");
    //printMatrix(image,width,height);
    gpu_analysis<<<grid_dim, block_dim>>>(width, height);
    //cudaErrorCheck(hipMemcpyFromArray(image, gpuImage, 0, 0,width*height*sizeof(int), hipMemcpyDeviceToHost));
    while(result == false) {
        //printf("AFTER ANALYSIS...\n");
        //printMatrix(image,width,height);
        gpu_link<<<grid_dim, block_dim>>>(width, height);
        //cudaErrorCheck(hipMemcpyFromArray(image, gpuImage, 0, 0,width*height*sizeof(int), hipMemcpyDeviceToHost));
        //printf("AFTER LINK...\n");
        //printMatrix(image,width,height);
        gpu_relabel<<<grid_dim, block_dim>>>(width, height);
        //cudaErrorCheck(hipMemcpyFromArray(image, gpuImage, 0, 0,width*height*sizeof(int), hipMemcpyDeviceToHost));
        //printf("AFTER RE-LABELLING...\n");
        //printMatrix(image,width,height);
        result = true;
        cudaErrorCheck(hipMemcpyToSymbol(HIP_SYMBOL(done), &result, sizeof(bool)));
        gpu_rescan<<<grid_dim, block_dim>>>(width, height);
        cudaErrorCheck(hipMemcpyFromSymbol(&result, HIP_SYMBOL(done), sizeof(bool)));
    }

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Time elapsed (gpu): %.6f ms\n",milliseconds);

    cudaErrorCheck(hipMemcpyFromArray(image, gpuImage, 0, 0,width*height*sizeof(int), hipMemcpyDeviceToHost));
    // apparently you don't have to unbind surfaces.
    cudaErrorCheck(hipFreeArray(gpuImage));
    
    //Colourise
    colourise(image,output,width,height);
}

int main(int argc, char **argv) {
    printf("%s Starting...\n\n", argv[0]);

    //initialize CUDA
    findCudaDevice(argc, (const char **)argv);

    //source and results image filenames
    char SampleImageFname[] = "3pixeldeath.bmp";
    char *pSampleImageFpath = sdkFindFilePath(SampleImageFname, argv[0]);

    if (pSampleImageFpath == NULL) {
        printf("%s could not locate Sample Image <%s>\nExiting...\n", pSampleImageFpath);
        exit(EXIT_FAILURE);
    }

    BMP input;
    BMP output;

	printf("===============================================\n");
    printf("Loading image: %s...\n", pSampleImageFpath);
    bool result = input.ReadFromFile(pSampleImageFpath);
	if (result == false) {
        printf("\nError: Image file not found or invalid!\n");
        exit(EXIT_FAILURE);
        return 1;
    }
    printf("===============================================\n");

	int width = input.TellWidth();
	int height = input.TellHeight();
	output.SetSize(width,height);
	output.SetBitDepth(32); // RGBA
    DataBlock   data;
    CPUBitmap bitmap( width, height, &data );
    data.bitmap = &bitmap;
    //HANDLE_ERROR( hipEventCreate( &data.start ) );
    //HANDLE_ERROR( hipEventCreate( &data.stop ) );
    copyBMPtoBitmap(&input,&bitmap);
    int* binaryImage = new int[width*height];
    bitmapToBinary(&bitmap,binaryImage);

    printf("LABELLING...\n");

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

   // double start_time = omp_get_wtime();

    gpu_label(binaryImage,&bitmap,width,height);

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("FINISHED...\n");
    //printf("Time elapsed: %f ms\n",(end_time-start_time)*1000.0);
    printf("Time elapsed (total): %.6f ms\n",milliseconds);

    copyBitmapToBMP(&bitmap,&output);
    //binaryToBitmap(binaryImage,&bitmap);
    //copyBitmapToBMP(&bitmap,&output);
    //HANDLE_ERROR( hipMemcpy( bitmap.get_ptr(), ImgSrc, imageSize, hipMemcpyHostToHost ) );
    //DumpBmpAsGray("out.bmp", ImgSrc, ImgStride, ImgSize);
    output.WriteToFile("out.bmp");
    bitmap.display_and_exit((void (*)(void*))anim_exit);
    delete[] binaryImage;
}

